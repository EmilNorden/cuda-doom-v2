#include "rt_entities.cuh"
#include "rt_raytracing.cuh"
#include "renderer/device_sprite.cuh"
#include "renderer/device_texture.cuh"
#include "renderer/cuda_utils.cuh"
#include "renderer/scene_entity.cuh"
#include "r_main.h"

namespace detail {
    std::array<std::optional<DeviceSprite>, NUMSPRITES> device_sprite_cache;
}

std::optional<DeviceSprite> RT_GetDeviceSprite(spritenum_t sprite);

SceneEntity *RT_CreateMapThing(mobjtype_t type, mobj_t *obj) {
    auto info = &mobjinfo[type];
    if(info->flags & MF_NOSECTOR) {
        // Is invisible
        return nullptr;
    }

    auto spawn_state = &states[info->spawnstate];

    auto sprite = RT_GetDeviceSprite(spawn_state->sprite);


    if (!sprite.has_value()) {
        std::cerr << "Couldnt find sprite " << spawn_state->sprite << "!\n";
        return nullptr;
    }

    auto frame = spawn_state->frame;
    auto position = glm::vec3(
            RT_FixedToFloating(obj->x),
            RT_FixedToFloating(obj->z),
            RT_FixedToFloating(obj->y));
    return create_device_type<SceneEntity>(position,
                                           frame & FF_FRAMEMASK,
                                           0,
                                           sprite.value());
}

void RT_DestroySceneEntity(SceneEntity *entity) {
    if (!entity) {
        return;
    }

    hipFree(entity);
}

void RT_UpdateEntityPosition(mobj_t *obj) {
    if (!obj->scene_entity) {
        return;
    }

    obj->scene_entity->position = glm::vec3{
            RT_FixedToFloating(obj->x),
            RT_FixedToFloating(obj->z),
            RT_FixedToFloating(obj->y)
    };

    obj->scene_entity->sprite = RT_GetDeviceSprite(obj->sprite).value();
    obj->scene_entity->frame = obj->frame & FF_FRAMEMASK;
    auto ang = R_PointToAngle(obj->x, obj->y);
    auto rot = (ang - obj->angle + (unsigned) (ANG45 / 2) * 9) >> 29;
    obj->scene_entity->rotation = rot;
}

std::optional<DeviceSprite> RT_GetDeviceSprite(spritenum_t s) {
    if(detail::device_sprite_cache[s].has_value()) {
        return detail::device_sprite_cache[s];
    }

    auto sprite = detail::sprite_data->sprites()[s];
    std::vector<DeviceSpriteFrame> device_sprite_frames;
    for (auto &sprite_frame: sprite.frames) {
        std::array<DeviceTexture *, 8> rotation_textures{};
        std::array<glm::i16vec2, 8> texture_offsets{};
        if (sprite_frame.rotate) {
            for (int rot = 0; rot < 8; ++rot) {
                auto sprite_lump = detail::sprite_data->sprite_lumps_start() +
                                   sprite_frame.lumps[rot]; // TODO: Store lump name in frame.lumps[] instead?
                auto sprite_lump_name = detail::wad->get_lump_name(sprite_lump);

                const auto &picture = detail::graphics_data->get_sprite(sprite_lump_name);

                texture_offsets[rot] = glm::i16vec2(picture.left_offset, picture.top_offset);

                if (sprite_frame.flip[rot]) {
                    auto flipped = wad::flip_picture(picture);
                    rotation_textures[rot] = create_device_type<DeviceTexture>(flipped.pixels, flipped.width,
                                                                               flipped.height);
                } else {
                    rotation_textures[rot] = create_device_type<DeviceTexture>(picture.pixels, picture.width,
                                                                               picture.height);
                }
            }
        } else {
            auto sprite_lump = detail::sprite_data->sprite_lumps_start() +
                               sprite_frame.lumps[0]; // TODO: Store lump name in frame.lumps[] instead?
            auto sprite_lump_name = detail::wad->get_lump_name(sprite_lump);

            const auto &picture = detail::graphics_data->get_sprite(sprite_lump_name);

            for (int rot = 0; rot < 8; ++rot) {
                rotation_textures[rot] = create_device_type<DeviceTexture>(picture.pixels, picture.width,
                                                                           picture.height);
                texture_offsets[rot] = glm::i16vec2(picture.left_offset, picture.top_offset);
            }
        }


        device_sprite_frames.emplace_back(rotation_textures, texture_offsets);
    }

    DeviceSprite device_sprite(device_sprite_frames);

    detail::device_sprite_cache[s] = device_sprite;

    return device_sprite;
}