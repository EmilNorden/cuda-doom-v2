#include "hip/hip_runtime.h"
#include "rt_raytracing.cuh"
#include <iostream>
#include <GL/glew.h>
#include "renderer/renderer.cuh"
#include "renderer/cuda_utils.cuh"
#include "renderer/camera.cuh"
#include "renderer/device_random.cuh"
#include "renderer/scene.cuh"
#include "rt_raytracing_opengl.cuh"
#include "rt_raytracing_scene.cuh"
#include "opengl/common.h"
#include "wad/graphics_data.cuh"
#include "wad/wad.cuh"
#include "wad/sprites.cuh"
#include "p_spec.h"
#include "rt_material.cuh"
#include <glm/gtx/rotate_vector.hpp>

// CUDA <-> OpenGL interop
namespace device {
    GLuint opengl_tex_cuda;
    Renderer *renderer;
    Camera *camera;
    Scene *scene;
    RandomGeneratorPool *random;
    std::uint8_t *palette;
}

namespace detail {
    wad::GraphicsData *graphics_data;
    wad::SpriteData *sprite_data;
    wad::Wad *wad;
    size_t current_sample;
    std::unordered_map<sector_t *, SectorGeometry> sector_geometry;
}


bool ray_tracing_enabled;

void print_cuda_device_info();

void init_gl_buffers();


void RT_InitGraphics(RayTracingInitOptions options) {
    std::vector<std::filesystem::path> paths;
    for (; *options.wadfiles; options.wadfiles++) {
        paths.emplace_back(*options.wadfiles);
    }

    detail::wad = new wad::Wad(paths);
    detail::graphics_data = new wad::GraphicsData(*detail::wad);
    detail::sprite_data = new wad::SpriteData(*detail::wad, sprnames, NUMSPRITES);
}


void RT_Init(RayTracingInitOptions options) {

    print_cuda_device_info();
    init_gl_buffers();

    hipDeviceSetLimit(hipLimitStackSize, 2048);

    detail::current_sample = 0;
    device::renderer = new Renderer(device::opengl_tex_cuda, 320, 240);
    device::camera = Camera::create();


    //auto camera_position = glm::vec3(-800.0, 20.0, -100.0);
    auto camera_position = glm::vec3(-765.315, 41.1001, -96.0371);// glm::vec3( -645.167, 58.7087, -412.004);
    auto camera_direction = glm::normalize(glm::vec3(0.0, 0.0, 0.0f) - camera_position);
    device::camera->set_position(camera_position);
    device::camera->set_direction(camera_direction);
    device::camera->set_up(glm::vec3(0.0, 1.0, 0.0));
    device::camera->set_field_of_view(75.0 * (3.1415 / 180.0));
    device::camera->set_blur_radius(0.0); // (0.03);
    device::camera->set_focal_length(60.0);
    device::camera->set_shutter_speed(0.0);
    device::camera->set_resolution(glm::vec2(320, 240));
    device::camera->update();
    std::cout << "Creating random states..." << std::flush;
    device::random = create_device_type<RandomGeneratorPool>(2048 * 256, 682856);
    std::cout << "Done." << std::endl;
    cuda_assert(hipMallocManaged(&device::palette, 768));

//std::vector<Square> &walls, std::vector<Triangle> &floors_ceilings, std::vector<MapThing> &map_things
    std::vector<Square *> walls;
    std::vector<Triangle *> fc;
    std::vector<SceneEntity *> mt;
    device::scene = create_device_type<Scene>(walls, fc, mt, nullptr);

    RT_InitGl();
    RT_InitGraphics(options);
    RT_InitMaterials(options);
}


void print_cuda_device_info() {
    int device_count = 0;
    hipError_t error_id = hipGetDeviceCount(&device_count);

    std::cout << "Using the following CUDA device: " << std::endl;

    if (error_id != hipSuccess) {
        std::cerr << "hipGetDeviceCount returned " << (int) error_id << "\n" << hipGetErrorString(error_id)
                  << std::endl;
        exit(1);
    }

    if (device_count == 0) {
        std::cout << "There are no available devices that support CUDA" << std::endl;
        exit(1);
    }

    int device_id = 0;

    hipSetDevice(device_id);
    hipDeviceProp_t device_properties{};
    hipGetDeviceProperties(&device_properties, device_id);

    std::cout << "  Name: " << device_properties.name << "\n";

    int driver_version, runtime_version;
    hipDriverGetVersion(&driver_version);
    hipRuntimeGetVersion(&runtime_version);

    printf("  CUDA Driver Version / Runtime Version %d.%d / %d.%d\n", driver_version / 1000,
           (driver_version % 100) / 10, runtime_version / 1000, (runtime_version % 100) / 10);
    printf("  CUDA Capability Major/Minor version number: %d.%d\n", device_properties.major,
           device_properties.minor);
    printf("  SM Count: %d, Warp size: %d, Shared mem/block %zu \n\n", device_properties.multiProcessorCount,
           device_properties.warpSize, device_properties.sharedMemPerBlock);

}

void init_gl_buffers() {
    const int WIDTH = 320;
    const int HEIGHT = 240;
    glGenTextures(1, &device::opengl_tex_cuda);
    glBindTexture(GL_TEXTURE_2D, device::opengl_tex_cuda);

    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, WIDTH, HEIGHT, 0, GL_RGBA, GL_FLOAT, NULL);

    check_for_gl_errors();
}

void RT_Enable() {
    ray_tracing_enabled = true;
}

void RT_Disable() {
    ray_tracing_enabled = false;
}

bool RT_IsEnabled() {
    return ray_tracing_enabled;
}

void RT_RenderSample() {
/*
    if(!detail::scene_entities_to_free.empty()) {
        for(auto entity : detail::scene_entities_to_free) {
            hipFree(entity);
        }

        detail::scene_entities_to_free.clear();
    }
*/
    device::renderer->render(
            device::camera,
            device::scene,
            device::random,
            device::palette,
            320,
            240,
            0);
    detail::current_sample++;
}

void RT_Present() {
    glActiveTexture(GL_TEXTURE0);
    glBindTexture(GL_TEXTURE_2D, device::opengl_tex_cuda);

    RT_RenderQuad();
}

void RT_UpdatePalette(byte *palette) {
    cuda_assert(hipMemcpy(device::palette, palette, 768, hipMemcpyHostToDevice));
}

void RT_BuildScene() {
    if (device::scene) {
        hipFree(device::scene);
        device::scene = nullptr;
    }

    auto result = RT_BuildScene(*detail::wad, *detail::graphics_data);
    device::scene = result.scene;
    detail::sector_geometry = result.sector_geometry;
}

void RT_UpdateCameraFromPlayer(player_t *player) {

    auto factor =
            static_cast<float>(player->mo->angle) / static_cast<float>(std::numeric_limits<unsigned>::max()) - 0.25f;
    auto radians = -factor * glm::two_pi<float>();
    auto direction = glm::normalize(glm::vec3(glm::sin(radians), 0, glm::cos(radians)));

    // Z is up
    device::camera->set_position({
                                         static_cast<float>(player->mo->x) / 65536.0f,
                                         static_cast<float>(player->viewz) / 65536.0f,
                                         static_cast<float>(player->mo->y) / 65536.0f});

    device::camera->set_direction(direction);
    if (device::camera->update()) {
        detail::current_sample = 0;
    }
}

void RT_WindowChanged() {
    init_gl_buffers();
    RT_InitGl();
    device::renderer = new Renderer(device::opengl_tex_cuda, 320, 240);
    printf("GL BUFFERS RECREATED\n");
}

void RT_AttachToScene(SceneEntity *entity) {
    if (!entity) {
        return;
    }
    //device::scene->add_entity(entity);

    //detail::has_pending_entities = true;
    //detail::pending_attach_entities.push_back(entity);
    device::scene->add_entity(entity);
}

void RT_DetachFromScene(SceneEntity *entity) {
    if (!entity) {
        return;
    }

    //detail::has_pending_entities = true;
    //detail::pending_detach_entities.push_back(entity);
    device::scene->remove_entity(entity);
}

void RT_VerticalDoorChanged(sector_t *sector) {
    auto it = detail::sector_geometry.find(sector);
    if (it == detail::sector_geometry.end()) {
        return;
    }

    auto &movable_sector = it->second;
    auto door = (vldoor_t *) sector->specialdata;

    auto ceiling_height = RT_FixedToFloating(sector->ceilingheight);
    auto door_total_height = RT_FixedToFloating(door->topheight) - RT_FixedToFloating(sector->floorheight);

    // Actual door
    for (auto wall: movable_sector.top_walls) {
        wall.wall->vertical_len = wall.adjacent_ceiling_height - ceiling_height;
        wall.wall->uv_offset = ceiling_height - RT_FixedToFloating(
                sector->floorheight); // door_total_height - (wall.adjacent_ceiling_height - ceiling_height);
    }

    // Side walls, ie door frame.
    for (auto wall: movable_sector.middle_walls) {
        wall->top_left.y = RT_FixedToFloating(door->topheight);
        wall->vertical_len = door_total_height;
        wall->vertical_vec = {0.0f, -1.0f, 0.0f};
        wall->uv_scale.y = (wall->vertical_len / wall->material.diffuse_map()->height()) / wall->vertical_len;
        if (wall->lower_unpegged) {
            wall->uv_offset = wall->material.diffuse_map()->height() - door_total_height;
        }

    }

    for (auto ceiling: movable_sector.ceiling) {
        ceiling->v0.y = ceiling->v1.y = ceiling->v2.y = ceiling_height;
    }
}

void RT_CeilingChanged(sector_t *sector) {
    auto it = detail::sector_geometry.find(sector);
    if (it == detail::sector_geometry.end()) {
        return;
    }

    auto &movable_sector = it->second;
    auto ceiling = (ceiling_t *) sector->specialdata;
    auto ceiling_height = RT_FixedToFloating(sector->ceilingheight);

    for (auto wall: movable_sector.top_walls) {
        wall.wall->top_left.y = glm::max(wall.adjacent_ceiling_height, ceiling_height);
        wall.wall->vertical_len = glm::abs(wall.adjacent_ceiling_height - ceiling_height);
        wall.wall->uv_offset = glm::abs(ceiling_height - RT_FixedToFloating(
                sector->floorheight)); // door_total_height - (wall.adjacent_ceiling_height - ceiling_height);
        // Dirty hack. Not sure if it'll work for all cases, but it certainly helps in map 2.
        if (wall.adjacent_ceiling_height < ceiling_height) {
            wall.wall->uv_offset -= glm::abs(wall.adjacent_ceiling_height - wall.wall->top_left.y) * 2;
        }
    }

    for (auto wall: movable_sector.adjacent_top_walls) {
        wall.wall->top_left.y = RT_FixedToFloating(ceiling->topheight);
        wall.wall->vertical_len = 100;
        wall.wall->uv_offset = ceiling_height - RT_FixedToFloating(
                sector->floorheight); // door_total_height - (wall.adjacent_ceiling_height - ceiling_height);
    }


    for (auto ceiling_tri: movable_sector.ceiling) {
        ceiling_tri->v0.y = ceiling_tri->v1.y = ceiling_tri->v2.y = ceiling_height;
    }
}

void RT_SectorFloorHeightChanged(sector_t *sector) {
    auto it = detail::sector_geometry.find(sector);
    if (it == detail::sector_geometry.end()) {
        return;
    }

    auto &movable_sector = it->second;

    auto floor_height = RT_FixedToFloating(sector->floorheight);
    for (auto wall: movable_sector.bottom_walls) {
        wall.wall->top_left.y = floor_height;

        wall.wall->vertical_len = glm::abs(wall.adjacent_floor_height - floor_height);
        wall.wall->vertical_vec = {0, -1, 0};
        wall.wall->uv_scale.y =
                (wall.wall->vertical_len / wall.wall->material.diffuse_map()->height()) / wall.wall->vertical_len;
    }

    for (auto wall: movable_sector.adjacent_bottom_walls) {
        wall.wall->top_left.y = glm::max(wall.adjacent_floor_height, floor_height);
        wall.wall->vertical_len = glm::length(wall.adjacent_floor_height - floor_height);
        wall.wall->vertical_vec = {0, -1, 0};
        wall.wall->uv_scale.y =
                (wall.wall->vertical_len / wall.wall->material.diffuse_map()->height()) / wall.wall->vertical_len;
    }


    for (auto wall: movable_sector.middle_walls) {
        wall->vertical_len = wall->top_left.y - floor_height;
        wall->vertical_vec = {0.0f, -1.0f, 0.0f};
        wall->uv_scale.y = (wall->vertical_len / wall->material.diffuse_map()->height()) / wall->vertical_len;
    }

    for (auto floor: movable_sector.floor) {
        floor->v0.y = floor_height;
        floor->v1.y = floor_height;
        floor->v2.y = floor_height;
    }
}